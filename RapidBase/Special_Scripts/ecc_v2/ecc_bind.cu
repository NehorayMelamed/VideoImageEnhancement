

#include "ecc_impl.cu"
#include <algorithm>
#include <array>
#include <bitset>
#include <chrono>
#include <climits>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <stdexcept>
#include <thrust/device_vector.h>
#include <thrust/functional.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/iterator/permutation_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/reduce.h>
#include <thrust/sequence.h>
#include <thrust/transform.h>
#include <torch/extension.h>
#include <torch/library.h>
#include <torch/torch.h>
#include <tuple>
#include <type_traits>
#include <vector>

#include <torch/extension.h>
#include <torch/library.h>
#include <torch/torch.h>
// #include <hip/hip_runtime.h>

// gx_chosen_values
// gy_chosen_values
// Jx
// Jy
// Jxx_prime
// Jxy_prime
// Jyx_prime
// Jyy_prime
// current_level_reference_tensor_zero_mean
// current_level_input_tensor_warped

using GLOB_TP = float;

auto ecc_reduction(const torch::Tensor &gx_chosen_values,
                   const torch::Tensor &gy_chosen_values,
                   const torch::Tensor &Jx,
                   const torch::Tensor &Jy,
                   const torch::Tensor &Jxx_prime,
                   const torch::Tensor &Jxy_prime,
                   const torch::Tensor &Jyx_prime,
                   const torch::Tensor &Jyy_prime,
                   const torch::Tensor &current_level_reference_tensor_zero_mean,
                   const torch::Tensor &current_level_input_tensor_warped)
{

  //  auto aaa= current_level_reference_tensor_zero_mean.data_ptr<GLOB_TP>();
  //  auto bbb= current_level_input_tensor_warped.data_ptr<GLOB_TP>();
  // std::cout << aaa << " " << thrust::reduce(thrust::device, aaa , aaa + 1000, GLOB_TP(-1), thrust::maximum<GLOB_TP>()) << std::endl;
  // std::cout << bbb << " " << thrust::reduce(thrust::device, bbb , bbb + 1000, GLOB_TP(-1), thrust::maximum<GLOB_TP>()) << std::endl;

  auto torch_input_sizes = gx_chosen_values.sizes();
  const std::array<int64_t, 2> input_size({*(torch_input_sizes.begin()), *(torch_input_sizes.end() - 1)});
  const std::array<const GLOB_TP *__restrict__, 9> inputs_ptr_arr({gx_chosen_values.data_ptr<GLOB_TP>(),
                                                                   gy_chosen_values.data_ptr<GLOB_TP>(),
                                                                   Jx.data_ptr<GLOB_TP>(),
                                                                   Jy.data_ptr<GLOB_TP>(),
                                                                   Jxx_prime.data_ptr<GLOB_TP>(),
                                                                   Jxy_prime.data_ptr<GLOB_TP>(),
                                                                   Jyx_prime.data_ptr<GLOB_TP>(),
                                                                   Jyy_prime.data_ptr<GLOB_TP>(),
                                                                   current_level_input_tensor_warped.data_ptr<GLOB_TP>()});

  const float *current_level_reference_tensor_zero_mean_ptr = current_level_reference_tensor_zero_mean.data_ptr<GLOB_TP>();

  ecc_inputs_ptr ecc_inputs(inputs_ptr_arr);

  auto output_tensor_options = // options for output tensor
      torch::TensorOptions()
          .dtype(Jx.dtype()) // Jx.dtype()
          .layout(torch::kStrided)
          .device(torch::kCUDA)
          .requires_grad(false); /* .device(torch::kCUDA, 0)  .dtype(torch::kInt32)*/

  torch::Tensor G = torch::empty({input_size[0], input_size[1], 8}, output_tensor_options);
  torch::Tensor Gt = torch::empty({input_size[0], 8,1}, output_tensor_options);
  torch::Tensor Gw = torch::empty({input_size[0], 8,1}, output_tensor_options);
  torch::Tensor C = torch::empty({input_size[0], 8, 8}, output_tensor_options);
  // torch::Tensor C = torch::empty({input_size[0], (8 + 1) * 8 / 2}, output_tensor_options);

  ecc_outputs_ptr ecc_outputs((Vec<GLOB_TP, 8> *__restrict__)(G.data_ptr<GLOB_TP>()),
                              (Vec<GLOB_TP, 8> *__restrict__)(Gt.data_ptr<GLOB_TP>()),
                              (Vec<GLOB_TP, 8> *__restrict__)(Gw.data_ptr<GLOB_TP>()),
                              (Mat<GLOB_TP, 8> *__restrict__)(C.data_ptr<GLOB_TP>()));

  // std::cout << input_size[0] << " " << input_size[1]  << std::endl;
  ecc_reduction_ker<<<input_size[0], 256>>>(ecc_inputs, current_level_reference_tensor_zero_mean_ptr, ecc_outputs, input_size);
  // ecc_reduction_impl<GLOB_TP>(inputs_ptr, outputs_ptr, input_size);  
  return std::make_tuple(G, Gt, Gw, C);
}

PYBIND11_MODULE(ecc_reduction, m)
{
  m.def("ecc_reduction", &ecc_reduction, "ecc_reduction");
}

// AT_DISPATCH_ALL_TYPES(Jx.dtype(), "ecc_reduction_impl", [&] { ecc_reduction_impl<scalar_t>(inputs, outputs); });
// AT_DISPATCH_GLOB_TPING_TYPES_AND_HALF(Jx.dtype(), "ecc_reduction_impl", [&] { ecc_reduction_impl<scalar_t>(inputs, outputs); });
// if ( Jx.dtype() ==  torch::GLOB_TP)
// {
//   ecc_reduction_impl<GLOB_TP>(inputs, outputs);
// }
// else if (Jx.dtype()== torch::dtype( torch::GLOB_TP))
// {
//   ecc_reduction_impl<GLOB_TP>(inputs, outputs);
// }
// else
// {
//   std::cout << "shit"  << std::endl;
// }