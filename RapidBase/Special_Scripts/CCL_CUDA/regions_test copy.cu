

#include <algorithm>
#include <array>
#include <bitset>
#include <chrono>
#include <climits>
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <stdexcept>
#include <torch/torch.h>
#include <vector>
#include <chrono>


//#include "conv_data_types.cu"
#include "regions_bind.cu"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


int main() {

  	using namespace std::chrono; 
    using namespace torch::indexing;	
	
	time_point<high_resolution_clock> start_point, end_point; // creating time points


  //torch::Tensor tensor = torch::ones({8000, 500,500});
  // auto in_sz = tensor.sizes();

  auto options =
      torch::TensorOptions()
          .dtype(torch::kBool)
          .layout(torch::kStrided)
          .device(torch::kCUDA)
          .requires_grad(false);


  int i,j;
  std::cout << "Please enter an integer value: ";
  std::cin >> i;
  std::cout << "Please enter an integer value: ";
  std::cin >> j;
  // torch::Tensor tst = torch::empty(tensor.sizes(),options);
  torch::Tensor tst = torch::ones({3, i,j}, options);
  //tst.index_put_({3,	Slice(), Slice()}, 0);
  //tst.index_put_({Slice(),	6, Slice()}, 0);
  //std::cout << tst.index({Slice(),	Slice(), 0}) ;
  tst.index_put_({0, Slice(), Slice(None, None,3)}, 0);
  tst.index_put_({0,	Slice(None, None,3), Slice()}, 0);

  tst.index_put_({1, Slice(), Slice(None, None,3)}, 0);
  
  tst.index_put_({2,	Slice(None, None,3), Slice()}, 0);


  //std::cout << "TST DATA PTR outer" << tst.data_ptr<bool>() << std::endl;

  //std::cout << "tick" << std::endl;
  //start_point = high_resolution_clock::now(); // storing the starting time point in start 
  auto CC_labeled_image = CCL(tst);
  //end_point = high_resolution_clock::now(); //storing the ending time in end 
  //std::cout << "tock" << std::endl;

  //auto start = time_point_cast<microseconds>(start_point).time_since_epoch().count(); 
	// casting the time point to microseconds and measuring the time since time epoch
	
	//auto end = time_point_cast<microseconds>(end_point).time_since_epoch().count();

  //std::cout << (end-start) << std::endl;
  //std::cout << tst.index({0,	Slice(), Slice()}) ;
  std::cout << CC_labeled_image.index({0,	Slice(), Slice()}) << std::endl;
  gpuErrchk(hipDeviceSynchronize());

  auto [blob_stats, n_blobs] = CC_Blob_Centers(CC_labeled_image);

  //std::cout << blob_stats.index({0,	Slice()}) << std::endl ;
  //std::cout << n_blobs << std::endl;
  //std::cout << n_blobs.sizes() << std::endl << n_blobs[0].sizes() <<std::endl;
  //std::cout << blob_stats.sizes() << std::endl << blob_stats[0].sizes() <<std::endl;
  //std::cout << blob_stats[0] << std::endl << n_blobs[0] << std::endl;
  for (int blob_num=0; blob_num < n_blobs[0].item<int>() ;blob_num++){
    std::cout <<  "blob x mean " <<  blob_stats[0][blob_num*3+0].item<float>()/float(blob_stats[0][blob_num*3+2].item<float>()) <<
     " " << "blob y mean " <<  blob_stats[0][blob_num*3+1].item<float>()/float(blob_stats[0][blob_num*3+2].item<float>()) << std::endl;
  }

    std::cout << CC_labeled_image.index({1,	Slice(), Slice()}) << std::endl;
  gpuErrchk(hipDeviceSynchronize());

  //std::cout << blob_stats.index({0,	Slice()}) << std::endl ;
  //std::cout << n_blobs << std::endl;
  //std::cout << n_blobs.sizes() << std::endl << n_blobs[0].sizes() <<std::endl;
  //std::cout << blob_stats.sizes() << std::endl << blob_stats[0].sizes() <<std::endl;
  //std::cout << blob_stats[0] << std::endl << n_blobs[0] << std::endl;
  for (int blob_num=0; blob_num < n_blobs[1].item<int>() ;blob_num++){
    std::cout <<  "blob x mean " <<  blob_stats[1][blob_num*3+0].item<float>()/float(blob_stats[1][blob_num*3+2].item<float>()) <<
     " " << "blob y mean " <<  blob_stats[1][blob_num*3+1].item<float>()/float(blob_stats[1][blob_num*3+2].item<float>()) << std::endl;
  }

      std::cout << CC_labeled_image.index({2,	Slice(), Slice()}) << std::endl;
  gpuErrchk(hipDeviceSynchronize());

  //std::cout << blob_stats.index({0,	Slice()}) << std::endl ;
  //std::cout << n_blobs << std::endl;
  //std::cout << n_blobs.sizes() << std::endl << n_blobs[0].sizes() <<std::endl;
  //std::cout << blob_stats.sizes() << std::endl << blob_stats[0].sizes() <<std::endl;
  //std::cout << blob_stats[0] << std::endl << n_blobs[0] << std::endl;
  for (int blob_num=0; blob_num < n_blobs[2].item<int>() ;blob_num++){
    std::cout <<  "blob x mean " <<  blob_stats[2][blob_num*3+0].item<float>()/float(blob_stats[2][blob_num*3+2].item<float>()) <<
     " " << "blob y mean " <<  blob_stats[2][blob_num*3+1].item<float>()/float(blob_stats[2][blob_num*3+2].item<float>()) << std::endl;
  }

  return 0;
};
