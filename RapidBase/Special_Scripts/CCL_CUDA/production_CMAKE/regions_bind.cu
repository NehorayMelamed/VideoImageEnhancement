

#include <algorithm>
#include <array>
#include <bitset>
#include <chrono>
#include <climits>
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <stdexcept>
#include <torch/torch.h>
#include <torch/extension.h>
#include <torch/library.h>
#include <vector>
#include <cstdio>
#include "CCL.cu"

using std::cout; using std::endl;
// #include "conv_data_types.cu"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

// void sup_dude() {
//   printf("sup dude priutnf");
//   cout << "sup dudeeeeeee" << endl;
// }

// void dummy(torch::Tensor & outliers_tensor) {
//   cout << "yo yo yo " << endl;
// }

auto CCL(torch::Tensor & outliers_tensor) {
  auto input_sizes = outliers_tensor.sizes();

  // cout << input_sizes << endl;
  //std::cout << input_sizes[0] << input_sizes[1] << input_sizes[2] << std::endl;
  auto output_tensor_options =
      torch::TensorOptions()
          .dtype(torch::kInt32)
          .layout(torch::kStrided)
          .device(torch::kCUDA, 0)
          .requires_grad(false);
 
  torch::Tensor CCL_labels_torch = torch::empty(outliers_tensor.sizes() , output_tensor_options);
  bool *input_tensor_raw_ptr = outliers_tensor.data_ptr<bool>();

  int *CCL_tensor_raw_ptr = CCL_labels_torch.data_ptr<int>();

#pragma unroll
  for (int z = 0; z < input_sizes[0]; z++) {
  connectedComponentLabeling( CCL_tensor_raw_ptr ,  input_tensor_raw_ptr , input_sizes[2], input_sizes[1]);
  //gpuErrchk(hipDeviceSynchronize());
  //cout << z << endl;
  CCL_tensor_raw_ptr += input_sizes[1]*input_sizes[2];
  input_tensor_raw_ptr += input_sizes[1]*input_sizes[2];
  }
  return CCL_labels_torch;
}

TORCH_LIBRARY(CCL, m) {
  m.def("CCL", &CCL);
 // m.def("dummy", &dummy, "dummy desc");
 // m.def("sup_dude", &sup_dude, "suo dude desccc");
}

